#include "hip/hip_runtime.h"
#include "gpufft.hpp"

#include <stdio.h>
#include <math.h>
#include <unordered_map>

struct fft_plan {
	int N;
	complex<float>* W;
	complex<float>* X;
	complex<float>* Y;
};

__global__ void fft_radix2_ct(fft_plan plan);

__global__ void fft_radix2(fft_plan plan) {
	const int& tid = threadIdx.x;
	const int& bsize = blockDim.x;
	const int& N = plan.N;
	const auto* const w = plan.W;
	auto* x0 = plan.X;
	auto* x1 = plan.Y;
	const int No2 = N >> 1;
	int s = No2;
	int s2 = N;
	while (((long) s2 * (long) s2) > N) {
		for (int i = tid; i < s; i += bsize) {
			int k = i;
			int j = i;
			for (int m = 0; m < No2; m += s) {
				const auto& z0 = x0[k];
				const auto z1 = x0[k + s] * w[m];
				x1[j] = z0 + z1;
				x1[j + No2] = z0 - z1;
				k += s2;
				j += s;
			}
		}
		s >>= 1;
		s2 >>= 1;
		auto* const tmp = x0;
		x0 = x1;
		x1 = tmp;
		__syncthreads();
	}
	while (s >= 1) {
		const int q = s2 * tid;
		const int a = s * tid;
		const int b = s * bsize;
		const int c = s2 * bsize;
		for (int i = 0; i < s; i++) {
			int k = i + q;
			int j = i + a;
			for (int m = s * tid; m < No2; m += b) {
				const auto& z0 = x0[k];
				const auto z1 = x0[k + s] * w[m];
				x1[j] = z0 + z1;
				x1[j + No2] = z0 - z1;
				k += c;
				j += b;
			}
		}
		s >>= 1;
		s2 >>= 1;
		auto* const tmp = x0;
		x0 = x1;
		x1 = tmp;
		__syncthreads();
	}
	if (plan.X != x0) {
		for (int k = tid; k < N; k += bsize) {
			plan.X[k] = x0[k];
		}
	}
	__syncthreads();
}

fft_plan fft_create_plan(int N) {
	fft_plan plan;
	plan.N = N;
	CUDA_CHECK(hipMallocManaged(&plan.W, sizeof(complex<float> ) * N));
	CUDA_CHECK(hipMallocManaged(&plan.X, sizeof(complex<float> ) * N));
	CUDA_CHECK(hipMallocManaged(&plan.Y, sizeof(complex<float> ) * N));
	if (plan.W == nullptr) {
		printf("Memory allocation failed %s %i\n", __FILE__, __LINE__);
	}
	if (plan.X == nullptr) {
		printf("Memory allocation failed %s %i\n", __FILE__, __LINE__);
	}
	if (plan.Y == nullptr) {
		printf("Memory allocation failed %s %i\n", __FILE__, __LINE__);
	}
	for (int n = 0; n < N; n++) {
		const float theta = -2.0 * M_PI * n / N;
		plan.W[n].real() = cos(theta);
		plan.W[n].imag() = sin(theta);
	}
	return plan;
}

void fft_destroy_plan(fft_plan plan) {
	CUDA_CHECK(hipFree(plan.W));
	CUDA_CHECK(hipFree(plan.X));
	CUDA_CHECK(hipFree(plan.Y));
}

int round_up(int n, int m) {
	return (((n - 1) / m) + 1) * m;
}

void gpufft(std::vector<complex<float>>& x) {
	const int N = x.size();
	const int nthreads = std::min(512, round_up(sqrt(N) * 2, 32));
	static std::unordered_map<int, fft_plan> plans;
	if (plans.find(N) == plans.end()) {
		plans[N] = fft_create_plan(N);
	}
	auto* y = plans[N].X;
	for (int n = 0; n < N; n++) {
		y[n] = x[n];
	}
	fft_radix2<<<1, nthreads>>>(plans[N]);
	CUDA_CHECK(hipDeviceSynchronize());
	for (int n = 0; n < N; n++) {
		x[n] = y[n];
	}
}

float rand1() {
	return 1.0 - 2.0 * (rand() + 0.5) / (RAND_MAX + 1.0);
}

int main(int argc, char **argv) {
	CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 16 * 1024));
	timer tm1;
	timer tm2;
	double t1 = 0.0;
	double t2 = 0.0;
	for (int N = 2; N < 128 * 1024 * 1024; N *= 2) {
		double err1 = 0.0;
		double err2 = 0.0;
		for (int i = 0; i < 8; i++) {
			std::vector<complex<float>> X(N);
			for (int n = 0; n < N; n++) {
				X[n].real() = rand1();
				X[n].imag() = rand1();
			}
			auto Y = X;
			auto X0 = X;
			tm2.start();
			fftw(Y);
			tm2.stop();
			tm1.start();
			gpufft(X);
			tm1.stop();
			//	t1 *= 2.0;
			if( i == 0) {
				tm1.reset();
				tm2.reset();
			}
			t1 += tm1.read();
			t2 += tm2.read();
			for (int n = 0; n < N; n++) {
				err1 = err1 + abs(X[n] - Y[n]);
				//	printf("%i : %e %e | %e %e\n", n, X[n].real(), X[n].imag(), Y[n].real(), Y[n].imag());
			}
		}
		printf("%i %e %e %e %e\n", N, err1 / N/8, t1, t2, t2 / t1);
	}
	return 0;
}
